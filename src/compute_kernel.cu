
#include <hip/hip_runtime.h>
extern "C"
__global__ void kernel_step_double(double *dot, double *kernel, double *vv_corr_rev, double * xdu_corr, double dt, int i, int len) {
    kernel[i] = 2. / vv_corr_rev[len - 1] * ((
            xdu_corr[i]
            - vv_corr_rev[len - 1 - i] * xdu_corr[0] / vv_corr_rev[len - 1]
        ) / dt - dot[0]);
}
